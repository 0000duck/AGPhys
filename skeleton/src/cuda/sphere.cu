#include "hip/hip_runtime.h"
//#define KINEMATIC
#define GRAVITY

#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include "hip/hip_vector_types.h"	// overload operators for floatN
#include "hip/hip_runtime_api.h"

#include "sphere.h"
#include "collision.h"

namespace CUDA {

__global__ void resetSpheresGrid(Sphere* spheres, int numberOfSpheres, int x, int z, float cornerX, float cornerY, float cornerZ, float distance)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < numberOfSpheres)
    {
        int layerSize = x * z;
        int yPos = tid / layerSize;
        int normId = tid - yPos * layerSize;

        int xPos = normId % x;
        int zPos = (normId - xPos) / x;

        spheres[tid].position.x = xPos * distance + cornerX;
        spheres[tid].position.y = yPos * distance + cornerY;
        spheres[tid].position.z = zPos * distance + cornerZ;
    }
}

__global__ void integrateSpheres(Sphere* spheres, int numberOfSpheres, float dt)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < numberOfSpheres)
    {
        Sphere& s = spheres[tid];

#ifdef GRAVITY
        s.impulse  += dt * make_float3(0, -1, 0); // gravity, breaks everything......
#endif
        s.newPos    = s.position + dt * s.impulse;

        // DEBUG
        s.color = make_float4(s.impulse) / 5 + make_float4(1, 1, 1, 0);
    }

}

__global__ void collideSpheres(Sphere* spheres, Plane* planes, int numberOfSpheres, int numberOfPlanes, float dt)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < numberOfSpheres)
    {
        Sphere& sphere = spheres[tid];

        IntersectionData firstIntersection = make_intersectiondata();

        // TEST PLANES
        for (int p = 0; p < numberOfPlanes; ++p)
        {
            Plane& plane = planes[p];

            IntersectionData currentIntersection = collideSpherePlane(&sphere, &plane, dt); // assumption: plane not moving
            if (currentIntersection.intersects)
            {
                if (!firstIntersection.intersects || currentIntersection.colTime < firstIntersection.colTime)
                {
                    firstIntersection = currentIntersection;
                }
            }
        }

        // TEST SPHERES
        for (int s = 0; s < numberOfSpheres; ++s)
        {
            if (s == tid) continue; // self

            Sphere& other = spheres[s];

            IntersectionData currentIntersection = collideSphereSphere(&sphere, &other, dt);
            if (currentIntersection.intersects)
            {
                if (!firstIntersection.intersects || currentIntersection.colTime < firstIntersection.colTime)
                {
                    firstIntersection = currentIntersection;
                }
            }
        }



        // RESOLVE COLLISION
        if (firstIntersection.intersects)
        {
#ifdef KINEMATIC
            resolveCollisionKinematically(&sphere, &firstIntersection);
#else
            resolveCollisionDynamically(&sphere, &firstIntersection);
#endif

        }
        sphere.position = sphere.newPos;

    }
}

__global__ void updateSpheres(Sphere* spheres, int numberOfSpheres)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < numberOfSpheres)
    {
        Sphere& sphere = spheres[tid];

        sphere.position = sphere.newPos;
    }
}

void resetSpheres(Sphere* spheres, int numberOfSpheres, int x, int z, float cornerX, float cornerY, float cornerZ, float distance)
{
    int threadsPerBlock = 128;
    int blocks = numberOfSpheres / threadsPerBlock + 1;
    resetSpheresGrid<<<blocks, threadsPerBlock>>>(spheres, numberOfSpheres, x, z, cornerX, cornerY, cornerZ, distance);
}

void updateAllSpheres(Sphere* spheres, Plane* planes, int numberOfSpheres, int numberOfPlanes, float dt)
{
    int threadsPerBlock = 128;
    int blocks = numberOfSpheres / threadsPerBlock + 1;
    integrateSpheres<<<blocks, threadsPerBlock>>>(spheres, numberOfSpheres, dt); // this way all threads are up to date
    collideSpheres<<<blocks, threadsPerBlock>>>(spheres, planes, numberOfSpheres, numberOfPlanes, dt);
    updateSpheres<<<blocks, threadsPerBlock>>>(spheres, numberOfSpheres);
}


}

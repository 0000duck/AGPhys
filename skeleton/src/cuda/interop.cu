#include "interop.h"


namespace 
{
#define checked_cuda(ans) { gpu_assert((ans), __FILE__, __LINE__); }
inline void gpu_assert(hipError_t code, char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}
}

void Interop::registerGLBuffer(int glbuffer)
{
	hipError_t err = hipGraphicsGLRegisterBuffer(&resource, glbuffer, hipGraphicsRegisterFlagsNone);
	checked_cuda(err);
}

void Interop::map()
{
	hipError_t err = hipGraphicsMapResources(1, &resource);
	checked_cuda(err);
}

void Interop::unmap()
{
	hipError_t err = hipGraphicsUnmapResources(1, &resource);
	checked_cuda(err);
}

void* Interop::getDevicePtr()
{
	void* ptr;
	size_t size;
	hipError_t err = hipGraphicsResourceGetMappedPointer(&ptr, &size, resource);
	checked_cuda(err);
	return ptr;
}

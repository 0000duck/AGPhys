#include "hip/hip_runtime.h"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include "hip/hip_vector_types.h"	// overload operators for floatN
#include "hip/hip_runtime_api.h"
#include "random.h"


#include "random.h"

#include "particle.h"

// Converts degrees to radians.
#define degreesToRadians(angleDegrees) (angleDegrees * 3.14159265359 / 180.0)
// Converts radians to degrees.
#define radiansToDegrees(angleRadians) (angleRadians * 180.0 / 3.14159265359)

using std::cout;
using std::endl;

namespace CUDA{


#define checked_cuda(ans) { gpu_assert((ans), __FILE__, __LINE__); }
inline void gpu_assert(hipError_t code, char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#define START_CUDA_TIMER hipEvent_t start, stop;\
    hipEventCreate(&start);hipEventCreate(&stop);\
    hipEventRecord(start);
#define STOP_CUDA_TIMER(TIME)  hipEventRecord(stop);hipEventSynchronize(stop);\
    hipEventElapsedTime(&TIME, start, stop);\
    hipEventDestroy(start); hipEventDestroy(stop);



__device__ unsigned int get_global_index(void)
{
    unsigned int threadid = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;
    unsigned int block_width =  blockDim.x * blockDim.y;
    unsigned int globalThreadNum = blockNumInGrid * block_width + threadid;
    return globalThreadNum;
}

<<<<<<< HEAD
__global__ void resetParticlesGrid(Particle* particles, int numberOfParticles, int x, int z, float cornerX, float cornerY, float cornerZ, float distance)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < numberOfParticles)
	{		
=======

__global__ void testKernel(float *g_idata, float *g_odata)
{
    int tid = threadIdx.x;

    // write data to global memory
    g_odata[tid] = g_idata[tid]+5;
}

void test()
{
    unsigned int num_threads = 32;
    unsigned int mem_size = sizeof(float) * num_threads;

    // allocate host memory
    float *h_idata = (float *) malloc(mem_size);

    // initalize the memory
    for (unsigned int i = 0; i < num_threads; ++i)
    {
        h_idata[i] = (float) i;
    }

    // allocate device memory
    float *d_idata;
    hipMalloc((void **) &d_idata, mem_size);
    // copy host memory to device
    hipMemcpy(d_idata, h_idata, mem_size,hipMemcpyHostToDevice);

    // allocate device memory for result
    float *d_odata;
    hipMalloc((void **) &d_odata, mem_size);

    // setup execution parameters
    dim3  grid(1, 1, 1);
    dim3  threads(num_threads, 1, 1);

    // execute the kernel
    testKernel<<< grid, threads, mem_size >>>(d_idata, d_odata);


    // allocate mem for the result on host side
    float *h_odata = (float *) malloc(mem_size);
    // copy result from device to host
    hipMemcpy(h_odata, d_odata, sizeof(float) * num_threads,hipMemcpyDeviceToHost);

    bool result = true;
    for (unsigned int i = 0; i < num_threads; ++i)
    {
        std::cout<<h_odata[i]<<std::endl;
        if(h_odata[i]!=i+5)
            result = false;
    }

    if(result){
        std::cout<<"CUDA test: SUCCESS!"<<std::endl;
    }else{
         std::cout<<"CUDA test: FAILED!"<<std::endl;
    }



    // cleanup memory
    free(h_idata);
    free(h_odata);
    hipFree(d_idata);
    hipFree(d_odata);

}

__global__ void resetParticlesGrid(Particle* particles, int numberOfParticles, int x, int z, float cornerX, float cornerY, float cornerZ, float distance)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < numberOfParticles)
	{
>>>>>>> cc15008e98bd5bd707b57c1c35f633ac156bf3f3
		int layerSize = x * z;
		int yPos = tid / layerSize;
		int normId = tid - yPos * layerSize;
		
		int xPos = normId % x;
		int zPos = (normId - xPos) / x;
		
		particles[tid].position.x = xPos * distance + cornerX;
		particles[tid].position.y = yPos * distance + cornerY;
		particles[tid].position.z = zPos * distance + cornerZ;
		
		particles[tid].color.x = 1.0;
		particles[tid].color.y = 0.0;
		particles[tid].color.z = 0.0;
	}
}

__global__ void resetParticlesVolcanoAndStar(Particle* particles, int numberOfParticles)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < numberOfParticles)
	{
		particles[tid].lifetime = 0;
		particles[tid].color	= make_float4(1);
	}
}

__device__ void integrateParticle(Particle* p, float dt)
{
	p->lifetime    -= dt;
	p->position	   += p->impulse * dt;
	p->color.x 		= p->lifetime / p->max_lifetime;
	p->color.y		= 0.0;
	p->color.z		= 1.0 - (p->lifetime / p->max_lifetime);
}

__global__ void integrateParticlesStar(Particle* particles, int numberOfParticles, float dt)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < numberOfParticles)
	{
		Particle* p 	= &particles[tid];
		integrateParticle(p, dt);
		
		if (p->lifetime < 0)
		{
			p->max_lifetime	= noise(7, 10);
			p->lifetime		= p->max_lifetime;
			
			p->position		= make_float3(0);
			p->impulse		= normalize(noise3D(make_float3(-1), make_float3(1)));
		}
	}
}

__global__ void integrateParticlesVolcano(Particle* particles, int numberOfParticles, float maxAngle, float dt)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;
	if (tid < numberOfParticles)
	{
		Particle* p 	= &particles[tid];
		integrateParticle(p, dt);
		
		if (p->lifetime < 0)
		{
			p->max_lifetime	= noise(7, 10);
			p->lifetime		= p->max_lifetime;
			
			p->position		= make_float3(0);
			float3 dir = make_float3(0); dir.y = 1; // upwards
			p->impulse		= normalize(distractDirection3D(dir, maxAngle));
		}
	}
}
<<<<<<< HEAD
=======


static const int threadsPerBlock = 128;
>>>>>>> cc15008e98bd5bd707b57c1c35f633ac156bf3f3

void resetParticlesGrid(void* particles, int numberOfParticles, int x, int z, float cornerX, float cornerY, float cornerZ, float distance)
{
	int blocks = numberOfParticles / threadsPerBlock + 1;
	resetParticlesGrid<<<blocks, threadsPerBlock>>>(static_cast<Particle*>(particles), numberOfParticles, x, z, cornerX, cornerY, cornerZ, distance);
}

void resetParticlesVolcanoAndStar(void* particles, int numberOfParticles)
{
	int blocks = numberOfParticles / threadsPerBlock + 1;
	resetParticlesVolcanoAndStar<<<blocks, threadsPerBlock>>>(static_cast<Particle*>(particles), numberOfParticles);
}

void integrateParticlesStar(void* particles, int numberOfParticles, float dt)
{
	int blocks = numberOfParticles / threadsPerBlock + 1;
	initRandom(numberOfParticles);
	
	integrateParticlesStar<<<blocks, threadsPerBlock>>>(static_cast<Particle*>(particles), numberOfParticles, dt);
}

void integrateParticlesVolcano(void* particles, int numberOfParticles, float maxAngle, float dt)
{
	int blocks = numberOfParticles / threadsPerBlock + 1;
	initRandom(numberOfParticles);
	
	integrateParticlesVolcano<<<blocks, threadsPerBlock>>>(static_cast<Particle*>(particles), numberOfParticles, degreesToRadians(maxAngle), dt);
}

}

#include "hip/hip_runtime.h"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include "hip/hip_vector_types.h"	// overload operators for floatN
#include "hip/hip_runtime_api.h"

#include "particle.h"

using std::cout;
using std::endl;

namespace CUDA{


#define checked_cuda(ans) { gpu_assert((ans), __FILE__, __LINE__); }
inline void gpu_assert(hipError_t code, char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

#define START_CUDA_TIMER hipEvent_t start, stop;\
    hipEventCreate(&start);hipEventCreate(&stop);\
    hipEventRecord(start);
#define STOP_CUDA_TIMER(TIME)  hipEventRecord(stop);hipEventSynchronize(stop);\
    hipEventElapsedTime(&TIME, start, stop);\
    hipEventDestroy(start); hipEventDestroy(stop);



__device__ unsigned int get_global_index(void)
{
    unsigned int threadid = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;
    unsigned int block_width =  blockDim.x * blockDim.y;
    unsigned int globalThreadNum = blockNumInGrid * block_width + threadid;
    return globalThreadNum;
}


__global__ void testKernel(float *g_idata, float *g_odata)
{
    int tid = threadIdx.x;

    // write data to global memory
    g_odata[tid] = g_idata[tid]+5;
}

void test()
{
    unsigned int num_threads = 32;
    unsigned int mem_size = sizeof(float) * num_threads;

    // allocate host memory
    float *h_idata = (float *) malloc(mem_size);

    // initalize the memory
    for (unsigned int i = 0; i < num_threads; ++i)
    {
        h_idata[i] = (float) i;
    }

    // allocate device memory
    float *d_idata;
    hipMalloc((void **) &d_idata, mem_size);
    // copy host memory to device
    hipMemcpy(d_idata, h_idata, mem_size,hipMemcpyHostToDevice);

    // allocate device memory for result
    float *d_odata;
    hipMalloc((void **) &d_odata, mem_size);

    // setup execution parameters
    dim3  grid(1, 1, 1);
    dim3  threads(num_threads, 1, 1);

    // execute the kernel
    testKernel<<< grid, threads, mem_size >>>(d_idata, d_odata);


    // allocate mem for the result on host side
    float *h_odata = (float *) malloc(mem_size);
    // copy result from device to host
    hipMemcpy(h_odata, d_odata, sizeof(float) * num_threads,hipMemcpyDeviceToHost);

    bool result = true;
    for (unsigned int i = 0; i < num_threads; ++i)
    {
        std::cout<<h_odata[i]<<std::endl;
        if(h_odata[i]!=i+5)
            result = false;
    }

    if(result){
        std::cout<<"CUDA test: SUCCESS!"<<std::endl;
    }else{
         std::cout<<"CUDA test: FAILED!"<<std::endl;
    }



    // cleanup memory
    free(h_idata);
    free(h_odata);
    hipFree(d_idata);
    hipFree(d_odata);

}

__device__ float generateRandomNumber(hiprandState* state, float min, float max)
{
	int tid = get_global_index();
	hiprandState localState = state[tid];
	float random = hiprand_uniform(&localState) * (max - min) + min;
	state[tid] = localState;
	return random;
}

__device__ float noise(hiprandState* state, float min, float max)
{
	return generateRandomNumber(state, min, max);
}

__device__ vec2_t noise2D(hiprandState* state, vec2_t min, vec2_t max)
{
	vec2_t result;
	result.x = generateRandomNumber(state, min.x, max.x);
	result.y = generateRandomNumber(state, min.y, max.y);
	return result;
}

__device__ vec3_t noise3D(hiprandState* state, vec3_t min, vec3_t max)
{
	vec3_t result;
	result.x = generateRandomNumber(state, min.x, max.x);
	result.y = generateRandomNumber(state, min.y, max.y);
	result.z = generateRandomNumber(state, min.z, max.z);
	return result;
}

__device__ vec3_t distractDirection3D(hiprandState* state, vec3_t origDirection, float maxAngle)
{
	float theta = noise(state, 0, maxAngle);
	float x = noise(state, 0, 0.99999999);
	float phi = 2 * 3.14159265359 * x;
	
	vec3_t yAxis = normalize(origDirection);
	vec3_t a; a.x = 1; a.y = 0; a.z = 0;
	if (yAxis.x == a.x && yAxis.y == a.y && yAxis.z == a.z)
	{
		a.x = 0; a.y = 1;
	}
	
	vec3_t xAxis = cross(yAxis, a);
	vec3_t zAxis = cross(xAxis, yAxis);
	
	
	vec3_t direction = sin(theta) * cos(phi) * zAxis + sin(theta) * sin(phi) * xAxis + cos(theta) * yAxis;
	return direction;
}

__global__ void resetParticles(Particle* particles, int numberOfParticles, int x, int z, float cornerX, float cornerY, float cornerZ, float distance)
{
	
	int tid = threadIdx.y * x * z + threadIdx.z * x + threadIdx.x;
	if (tid < numberOfParticles)
	{
		particles[tid].initial_position.x = threadIdx.x * distance + cornerX;
		particles[tid].initial_position.y = threadIdx.y * distance + cornerY;
		particles[tid].initial_position.z = threadIdx.z * distance + cornerZ;
		
		particles[tid].initial_color.x = 1.0;
		particles[tid].initial_color.y = 0.0;
		particles[tid].initial_color.z = 0.0;
		particles[tid].initial_color.w = 1.0;
		
		particles[tid].position = particles[tid].initial_position;
		particles[tid].color	= particles[tid].initial_color;
		
		particles[tid].impulse.x = 0;
		particles[tid].impulse.y = 0.1;
		particles[tid].impulse.z = 0;
		
		particles[tid].max_lifetime = 100;
		particles[tid].lifetime = particles[tid].max_lifetime;
	}
}

__device__ void integrateSingleParticle(Particle* particle, float dt, hiprandState* state)
{
	particle->position += dt * particle->impulse;
	particle->lifetime -= dt;
	particle->color.z = particle->lifetime / particle->max_lifetime;
	
	if (particle->lifetime < 0)
	{
		vec3_t min; min.x = 0; min.y = 0; min.z = 0;
		vec3_t max; max.x = 15; max.y = 15; max.z = 15;
		//particle->position 	= noise3D(state, min, max);
		particle->position 	= make_float3(0);
		particle->color 	= particle->initial_color;
		
		vec3_t impulse; impulse.x = 1; impulse.y = 1; impulse.z = 0;
		particle->impulse = distractDirection3D(state, impulse, 0.12);
		
		particle->max_lifetime 	= noise(state, 100, 115);
		particle->lifetime		= particle->max_lifetime;
	}
}

__global__ void integrateParticles(Particle* particles, float dt, hiprandState* state)
{
	int tid = threadIdx.x;
	integrateSingleParticle(&particles[tid], dt, state);
}

__global__ void setupRandomNumberState(hiprandState* state, unsigned long seed)
{
    int tid = threadIdx.x;
    hiprand_init(seed, tid, 0, &state[tid]);
} 

static hiprandState* buildStates(int numberOfStates)
{
	hiprandState* state;
	hipMalloc((void **) &state, numberOfStates * sizeof(hiprandState));
	setupRandomNumberState<<<1, numberOfStates>>>(state, 1234);
	return state;
}

static void freeStates(hiprandState* state)
{
	hipFree(state);
}

void resetParticles(void* particles, int numberOfParticles, int x, int z, float cornerX, float cornerY, float cornerZ, float distance)
{
	int y = numberOfParticles / (x * z);
	if (numberOfParticles % (x * z) > 0) y++;
    dim3 threads(x, y, z);
	resetParticles<<<1, threads>>>(static_cast<Particle*>(particles), numberOfParticles, x, z, cornerX, cornerY, cornerZ, distance);
}

void integrateParticles(void* particles, int numberOfParticles, float dt)
{
	hiprandState* state = buildStates(numberOfParticles);
	
    dim3 threads(numberOfParticles, 1, 1);
    integrateParticles<<<1, threads>>>(static_cast<Particle*>(particles), dt, state);
    
    freeStates(state);
}

}

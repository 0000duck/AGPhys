#include "hip/hip_runtime.h"
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <iostream>
#include <ctime>
#include "hip/hip_vector_types.h"	// overload operators for floatN
#include "hip/hip_runtime_api.h"

// thrust
#include <thrust/device_vector.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/sort.h>
#include <thrust/reduce.h>

#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/quaternion.hpp>


#include "rigidbody.h"
#include "collision.h"

namespace CUDA
{

RigidBody* body_ptr;
int* grid_ptr;
int numberOfBodies;
int numberOfPlanes;

void initRigidBodies(RigidBody* host_bodies, int size, int planeCount)
{
    hipMalloc(&body_ptr, sizeof(RigidBody) * size);
    hipMemcpy(body_ptr, host_bodies, sizeof(RigidBody) * size, hipMemcpyHostToDevice);
    numberOfBodies = size;
    numberOfPlanes = planeCount;

    // init grid
    int gridSize = numberOfBodies * (numberOfBodies + numberOfPlanes);
    hipMalloc(&grid_ptr, sizeof(int) * gridSize);
}

void shutdownRigidBodies()
{
    hipFree(body_ptr);
    hipFree(grid_ptr);
}

__global__ void getPosAndRot(RigidBody* bodies, vec3_t* pos_ptr, quat_t* rot_ptr, int numberOfBodies)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < numberOfBodies)
    {
        RigidBody& rb = bodies[tid];
        pos_ptr[tid] = rb.position;
        rot_ptr[tid] = rb.rotation;
    }
}

void getOrientationData(std::vector<glm::vec3>& pos, std::vector<glm::quat>& rot)
{
    int threadsPerBlock = 128;
    int blocks = numberOfBodies / threadsPerBlock + 1;

    vec3_t* pos_ptr;
    quat_t* rot_ptr;

    hipMalloc(&pos_ptr, sizeof(vec3_t) * numberOfBodies);
    hipMalloc(&rot_ptr, sizeof(quat_t) * numberOfBodies);

    getPosAndRot<<<blocks, threadsPerBlock>>>(body_ptr, pos_ptr, rot_ptr, numberOfBodies);

    hipMemcpy(&pos[0], pos_ptr, sizeof(vec3_t) * numberOfBodies, hipMemcpyDeviceToHost);
    hipMemcpy(&rot[0], rot_ptr, sizeof(quat_t) * numberOfBodies, hipMemcpyDeviceToHost);

    hipFree(pos_ptr);
    hipFree(rot_ptr);
}

__host__ __device__ void quatToRot3(quat_t& quat, float rot[3][3])
{
    rot[0][0] = 1.f - 2.f * quat.y * quat.y - 2.f * quat.z * quat.z;
    rot[0][1] = 2.f * quat.x * quat.y - 2.f * quat.w * quat.z;
    rot[0][2] = 2.f * quat.x * quat.z + 2.f * quat.w * quat.y;

    rot[1][0] = 2.f * quat.x * quat.y + 2.f * quat.w * quat.z;
    rot[1][1] = 1.f - 2.f * quat.x * quat.x - 2.f * quat.z * quat.z;
    rot[1][2] = 2.f * quat.y * quat.z - 2.f * quat.w * quat.x;

    rot[2][0] = 2.f * quat.x * quat.z - 2.f * quat.w * quat.z;
    rot[2][1] = 2.f * quat.y * quat.z + 2.f * quat.w * quat.x;
    rot[2][2] = 1.f - 2.f * quat.x * quat.x - 2.f * quat.y * quat.y;
}

__host__ __device__ void transposeMatrix(float in[3][3], float out[3][3])
{
    for (int y = 0; y < 3; ++y)
    {
        for (int x = 0; x < 3; ++x)
        {
            out[x][y] = in[y][x];
        }
    }
}

__host__ __device__ void matTimesMat(float l[3][3], float r[3][3], float out[3][3])
{
    for (int row = 0; row < 3; ++row)
    {
        for (int col = 0; col < 3; ++col)
        {
            float sum = 0.f;
            for (int k = 0; k < 3; ++k)
            {
                sum += l[row][k] * r[k][col];
            }
            out[row][col] = sum;
        }
    }
}

__host__ __device__ void matTimesScalar(float m[3][3], float s)
{
    for (int y = 0; y < 3; ++y)
    {
        for (int x = 0; x < 3; ++x)
        {
            m[y][x] *= s;
        }
    }
}

__host__ __device__ void matTimesVec(float m[3][3], vec3_t& in, vec3_t& out)
{
    out.x = m[0][0] * in.x + m[0][1] * in.y + m[0][2] * in.z;
    out.y = m[1][0] * in.x + m[1][1] * in.y + m[1][2] * in.z;
    out.z = m[2][0] * in.x + m[2][1] * in.y + m[2][2] * in.z;
}

__host__ __device__ void quatTimesQuat(quat_t& l, quat_t& r, quat_t& out)
{
    float3 v0 = make_float3(l.x, l.y, l.z);
    float3 v1 = make_float3(r.x, r.y, r.z);

    out.w = l.w * r.w - dot(v0, v1);
    float3 v = l.w * v1 + r.w * v0 + cross(v0, v1);
    out.x = v.x;
    out.y = v.y;
    out.z = v.z;
}

__global__ void updateBodies(RigidBody* bodies, int numberOfBodies, float dt)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < numberOfBodies)
    {
        RigidBody& rb = bodies[tid];

        rb.position += (rb.linearVelocity / rb.mass) * dt;
        rb.linearVelocity += dt * make_float3(0.0, -1.5f, 0.0f); // gravity


        if (length(rb.angularMomentum) == 0.f)
            return;

        float rot[3][3];
        quatToRot3(rb.rotation, rot);
        float invRot[3][3];
        transposeMatrix(rot, invRot);

        // equation 6
        float curInertia[3][3];
        float tmp[3][3];
        matTimesMat(rot, rb.invInertia, tmp);
        matTimesMat(tmp, invRot, curInertia);

        // equation 5
        matTimesVec(curInertia, rb.angularMomentum, rb.angularVelocity);
        
        // equation 7
        float3 rotationAxis = normalize(rb.angularVelocity);
        float rotationAngle = length(rb.angularVelocity * dt);
        quaternion<float> dq(rotationAxis, rotationAngle);

        // equation 8
        quaternion<float> newRot = dq * rb.rotation;
        rb.rotation = newRot;
    }
}

__device__ inline void incrementGrid(int* grid, int width, int ownID, int otherID)
{
    int index = width * ownID + otherID;
    atomicAdd(&grid[index], 1);
}

__device__ inline void getAbsPositionAndVelocity(RigidBody& rb, Sphere& sphere, float3& pos, float3& vel)
{
    float3 abs_pos;
    float rot[3][3];
    quatToRot3(rb.rotation, rot);
    matTimesVec(rot, sphere.position, abs_pos);
    float3 ang = cross(rb.angularVelocity, abs_pos);
    float3 lin = rb.linearVelocity;
    vel = ang + lin;
    pos = abs_pos + rb.position;
}

__global__ void collisionDetection(RigidBody* bodies, int numberOfBodies, Sphere* spheres, int numberOfSpheres, Plane* planes, int numberOfPlanes, int* grid)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < numberOfSpheres)
    {
        int rb_ID = 0;
        int gridWidth = numberOfBodies + numberOfPlanes;

        uint sphereSum = 0;
        for (int r = 0; r < numberOfBodies; ++r)
        {
            sphereSum += bodies[r].numberOfSpheres;
            if (tid < sphereSum)
            {
                // found corresponding body
                rb_ID = r;
                break;
            }
        }


        Sphere& sphere = spheres[tid];
        RigidBody& rb = bodies[rb_ID];

        sphere.force = make_float3(0.0);

        // absolute position & velocity
        float3 abs_pos, abs_vel;
        getAbsPositionAndVelocity(rb, sphere, abs_pos, abs_vel);

        Sphere s = sphere;
        s.position = abs_pos;
        s.velocity = abs_vel;

        sphere.sphereCollider = -1;
        sphere.planeCollider  = -1;

        // PLANE COLLISION
        for (int p = 0; p < numberOfPlanes; ++p)
        {
            float penetration = collideSpherePlane(s, planes[p]);
            if (penetration != -1.0f)
            {
                incrementGrid(grid, gridWidth, rb_ID, numberOfBodies + p);
                sphere.planeCollider = p;
                rb.linearVelocity = make_float3(0.f);
                rb.angularMomentum = make_float3(0.f);
            }
        }

        // SPHERE COLLISION - brute force for now
        for (int s = 0; s < numberOfSpheres; ++s)
        {

        }




    }
}

__device__ inline void atomicAddAngularMomentum(RigidBody& rb, const float3& t)
{
    atomicAdd(&rb.angularMomentum.x, t.x);
    atomicAdd(&rb.angularMomentum.y, t.y);
    atomicAdd(&rb.angularMomentum.z, t.z);
}

__device__ inline void atomicAddLinearVelocity(RigidBody& rb, const float3& l)
{
    atomicAdd(&rb.linearVelocity.x, l.x);
    atomicAdd(&rb.linearVelocity.y, l.y);
    atomicAdd(&rb.linearVelocity.z, l.z);
}

__global__ void collisionResponse(RigidBody* bodies, int numberOfBodies, Sphere* spheres, int numberOfSpheres, Plane* planes, int numberOfPlanes, int* grid, float dt)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < numberOfSpheres)
    {
        int rb_ID = 0;
        int gridWidth = numberOfBodies + numberOfPlanes;

        uint sphereSum = 0;
        for (int r = 0; r < numberOfBodies; ++r)
        {
            sphereSum += bodies[r].numberOfSpheres;
            if (tid < sphereSum)
            {
                rb_ID = r;
                break;
            }
        }

        Sphere& sphere = spheres[tid];
        RigidBody& rb = bodies[rb_ID];

        // absolute position & velocity
        float3 abs_pos, abs_vel;
        getAbsPositionAndVelocity(rb, sphere, abs_pos, abs_vel);

        if (sphere.planeCollider != -1)
        {
            // PLANE RESPONSE

            Plane& plane = planes[sphere.planeCollider];
            int numberOfCollisions = grid[rb_ID * gridWidth + numberOfBodies + sphere.planeCollider];
            if (numberOfCollisions != 0)
            {
                float mass = rb.mass / numberOfCollisions;
                float vNormal = -length(dot(abs_vel,plane.normal) * plane.normal);
                float epsilon = 0.1;
                float j = -(1+epsilon) * mass *vNormal;



                float3 J = j * plane.normal;


                float l = length(abs_vel - (vNormal * plane.normal));
                if (l > 0){
                    //friction
                    float mu = 0.1;
                    float3 frictionTerm = (abs_vel - (vNormal * plane.normal)) / l;

                    J -= mu * j * frictionTerm;

                }

                const float deltaT = 1.0f;
                float3 momentum = J*deltaT;

                float3 toAdd = momentum ;

    //            particle.velocity += toAdd;

                sphere.force += toAdd;
            }
        }

        if (sphere.sphereCollider != -1)
        {
            // SPHERE RESPONSE
        }

        float3 linearForce = sphere.force;
        float3 torque = 0.1*cross(sphere.position, linearForce);


        //TODO use reduce
        atomicAddLinearVelocity(rb, linearForce);
        atomicAddAngularMomentum(rb, torque);

    }
}

__global__ void clearGrid(int* grid, int sizeOfGrid)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < sizeOfGrid)
    {
        grid[tid] = 0;
    }
}

void printVec(vec3_t& v)
{
    std::cout << "Vector: " << v.x << " " << v.y << " " << v.z << std::endl;
}

void printQuat(quat_t& q)
{
    std::cout << "Quaternion: " << q.x << " " << q.y << " " << q.z << " " << q.w << std::endl;
}

void printMat(float m[3][3])
{
    std::cout << "Matrix: " << std::endl;
    for (int y = 0; y < 3; ++y)
    {
        for (int x = 0; x < 3; ++x)
        {
            std::cout << m[y][x] << "    ";
        }
        std::cout << std::endl;
    }
}

void printDelim()
{
    std::cout << "-----------" << std::endl;
}

void updateRigidBodies(Sphere* spheres, int numberOfSpheres, Plane* planes, int numberOfPlanes, float dt)
{
    int threadsPerBlock = 128;
    int blocks = numberOfBodies / threadsPerBlock + 1;
    updateBodies<<<blocks, threadsPerBlock>>>(body_ptr, numberOfBodies, dt);
    blocks = (numberOfBodies * (numberOfBodies + numberOfPlanes)) / threadsPerBlock + 1;
    clearGrid<<<blocks, threadsPerBlock>>>(grid_ptr, numberOfBodies * (numberOfBodies + numberOfPlanes));
    blocks = numberOfSpheres / threadsPerBlock + 1;
    collisionDetection<<<blocks, threadsPerBlock>>>(body_ptr, numberOfBodies, spheres, numberOfSpheres, planes, numberOfPlanes, grid_ptr);
    collisionResponse<<<blocks, threadsPerBlock>>>(body_ptr, numberOfBodies, spheres, numberOfSpheres, planes, numberOfPlanes, grid_ptr, dt);

}



}

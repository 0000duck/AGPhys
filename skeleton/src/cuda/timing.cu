#include "timing.h"


#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>
#include "hip/hip_vector_types.h"
#include "hip/hip_runtime_api.h"

namespace
{
    #define checked_cuda(ans) { gpu_assert((ans), __FILE__, __LINE__); }
    inline void gpu_assert(hipError_t code, char *file, int line, bool abort=true) {
        if (code != hipSuccess) {
            fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
            if (abort) exit(code);
        }
    }
}

static hipEvent_t start = NULL;
static hipEvent_t stop = NULL;
static bool initialized = false;

void initTiming()
{
    if (initialized)
        return;

    hipError_t err = hipEventCreate(&start);
    checked_cuda(err);

    err = hipEventCreate(&stop);
    checked_cuda(err);

    initialized = true;
}

void shutdownTiming()
{
    if (!initialized)
        return;

    hipError_t err = hipEventDestroy(start);
    checked_cuda(err);

    err = hipEventDestroy(stop);
    checked_cuda(err);

    initialized = false;
}

void startTiming()
{
    hipError_t err = hipEventRecord(start);
    checked_cuda(err);
}

float endTiming()
{
    hipError_t err = hipEventRecord(stop);
    checked_cuda(err);

    hipEventSynchronize(stop);
    checked_cuda(err);

    float time = 0.0f;
    err = hipEventElapsedTime(&time, start, stop);
    checked_cuda(err);

    return time;
}
